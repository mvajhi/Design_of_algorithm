
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <ctime> 
using namespace std;

void generateRandomInput(vector<int>& weights, vector<int>& profits, int n, int maxWeight, int maxProfit) {
    for (int i = 0; i < n; i++) {
        weights.push_back(1 + rand() % maxWeight); // وزن تصادفی بین 1 و maxWeight
        profits.push_back(1 + rand() % maxProfit); // سود تصادفی بین 1 و maxProfit
    }
}

// Function to find the maximum profit
int knapSack(int W, vector<int> wt, vector<int> val) {
  
    // Making and initializing dp vector
    vector<int> dp(W + 1, 0);
    vector<int> dp_pre(W + 1, 0);

    for (int i = 1; i <= wt.size(); i++) {
        for (int w = W; w >= 0; w--) {
            if (wt[i - 1] <= w)
              
                // Finding the maximum value
                dp[w] = max(dp_pre[w], dp_pre[w - wt[i - 1]] + val[i - 1]);
        }
        dp_pre = dp;
    }
    return dp[W];
}

__global__ void knapSackKernel_dpx(int W, int n, int* wt, int* val, int* dp_pre, int* dp, int i) {
    int w = threadIdx.x + blockIdx.x * blockDim.x; // محاسبه اندیس وزن
    bool t;

    if (w <= W) {
        if (wt[i - 1] <= w) {
            dp[w] = __vibmax_u32(dp_pre[w], dp_pre[w - wt[i - 1]] + val[i - 1], &t);
        } else {
            dp[w] = dp_pre[w];
        }
    }
}

int knapSackCUDA_dpx(int W, vector<int>& wt, vector<int>& val) {
    int n = wt.size();
    int *d_wt, *d_val, *d_dp, *d_dp_pre;

    // تخصیص حافظه روی دستگاه
    hipMalloc(&d_wt, n * sizeof(int));
    hipMalloc(&d_val, n * sizeof(int));
    hipMalloc(&d_dp, (W + 1) * sizeof(int));
    hipMalloc(&d_dp_pre, (W + 1) * sizeof(int));

    // کپی داده‌ها از میزبان به دستگاه
    hipMemcpy(d_wt, wt.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, val.data(), n * sizeof(int), hipMemcpyHostToDevice);

    // مقداردهی اولیه به dp
    hipMemset(d_dp, 0, (W + 1) * sizeof(int));
    hipMemset(d_dp_pre, 0, (W + 1) * sizeof(int));

    int threadsPerBlock = 256;
    int numBlocks = (W + threadsPerBlock - 1) / threadsPerBlock;

    clock_t start_gpu = clock();

    // اجرای حلقه‌های جداگانه برای هر آیتم
    for (int i = 1; i <= n; i++) {
        knapSackKernel_dpx<<<numBlocks, threadsPerBlock>>>(W, n, d_wt, d_val, d_dp_pre, d_dp, i);
        hipDeviceSynchronize(); // اطمینان از تکمیل تمام بلاک‌ها

        // کپی dp به dp_pre برای مرحله بعدی
        hipMemcpy(d_dp_pre, d_dp, (W + 1) * sizeof(int), hipMemcpyDeviceToDevice);
    }

    clock_t end_gpu = clock();

    // کپی نتیجه نهایی dp[W]
    int result = 0;
    hipMemcpy(&result, &d_dp[W], sizeof(int), hipMemcpyDeviceToHost);

    double gpu_time = 1000.0 * (end_gpu - start_gpu) / CLOCKS_PER_SEC;
    cout << "GPU dpx Result: " << result << ", Time: " << gpu_time << " ms" << endl;

    // آزادسازی حافظه دستگاه
    hipFree(d_wt);
    hipFree(d_val);
    hipFree(d_dp);
    hipFree(d_dp_pre);

    return result;
}

__global__ void knapSackKernel(int W, int n, int* wt, int* val, int* dp_pre, int* dp, int i) {
    int w = threadIdx.x + blockIdx.x * blockDim.x; // محاسبه اندیس وزن

    if (w <= W) {
        if (wt[i - 1] <= w) {
            dp[w] = max(dp_pre[w], dp_pre[w - wt[i - 1]] + val[i - 1]);
        } else {
            dp[w] = dp_pre[w];
        }
    }
}

int knapSackCUDA(int W, vector<int>& wt, vector<int>& val) {
    int n = wt.size();
    int *d_wt, *d_val, *d_dp, *d_dp_pre;

    // تخصیص حافظه روی دستگاه
    hipMalloc(&d_wt, n * sizeof(int));
    hipMalloc(&d_val, n * sizeof(int));
    hipMalloc(&d_dp, (W + 1) * sizeof(int));
    hipMalloc(&d_dp_pre, (W + 1) * sizeof(int));

    // کپی داده‌ها از میزبان به دستگاه
    hipMemcpy(d_wt, wt.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, val.data(), n * sizeof(int), hipMemcpyHostToDevice);

    // مقداردهی اولیه به dp
    hipMemset(d_dp, 0, (W + 1) * sizeof(int));
    hipMemset(d_dp_pre, 0, (W + 1) * sizeof(int));

    int threadsPerBlock = 256;
    int numBlocks = (W + threadsPerBlock - 1) / threadsPerBlock;

    clock_t start_gpu = clock();

    // اجرای حلقه‌های جداگانه برای هر آیتم
    for (int i = 1; i <= n; i++) {
        knapSackKernel<<<numBlocks, threadsPerBlock>>>(W, n, d_wt, d_val, d_dp_pre, d_dp, i);
        hipDeviceSynchronize(); // اطمینان از تکمیل تمام بلاک‌ها

        // کپی dp به dp_pre برای مرحله بعدی
        hipMemcpy(d_dp_pre, d_dp, (W + 1) * sizeof(int), hipMemcpyDeviceToDevice);
    }

    clock_t end_gpu = clock();

    // کپی نتیجه نهایی dp[W]
    int result = 0;
    hipMemcpy(&result, &d_dp[W], sizeof(int), hipMemcpyDeviceToHost);

    double gpu_time = 1000.0 * (end_gpu - start_gpu) / CLOCKS_PER_SEC;
    cout << "GPU Result: " << result << ", Time: " << gpu_time << " ms" << endl;

    // آزادسازی حافظه دستگاه
    hipFree(d_wt);
    hipFree(d_val);
    hipFree(d_dp);
    hipFree(d_dp_pre);

    return result;
}

int main() {
     auto seed = time(0);
    // auto seed = 1733565654;
    cout << "Random seed: " << seed << endl;
    srand(seed); // مقداردهی اولیه برای تولید اعداد تصادفی

    int n = 2000; // تعداد آیتم‌ها
    int W = 100000; // ظرفیت کوله‌پشتی
    int maxWeight = 50; // بیشترین وزن ممکن
    int maxProfit = 100; // بیشترین سود ممکن

    vector<int> weights;
    vector<int> profits;

    // تولید ورودی تصادفی
    generateRandomInput(weights, profits, n, maxWeight, maxProfit);

    // اندازه‌گیری زمان اجرای CPU
    clock_t start_cpu = clock();
    int cpu_result = knapSack(W, weights, profits);
    clock_t end_cpu = clock();
    double cpu_time = 1000.0 * (end_cpu - start_cpu) / CLOCKS_PER_SEC;

    // اندازه‌گیری زمان اجرای GPU
    int gpu_dpx_result = knapSackCUDA_dpx(W, weights, profits);
    int gpu_result = knapSackCUDA(W, weights, profits);

    // نتایج و مقایسه زمان
    cout << "CPU Result: " << cpu_result << ", Time: " << cpu_time << " ms" << endl;

    return 0;
}